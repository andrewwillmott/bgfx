#ifndef __CUDA_NOISE_KERNEL_H
#define __CUDA_NOISE_KERNEL_H

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void calculateNoise(uint32_t width, uint32_t height, int seed, hipSurfaceObject_t image)
{
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
  {
    return;
  }

  int id = y * width + x;
  hiprandState rngState;
  hiprand_init(seed, id, 0, &rngState);

  uchar4 rgba;
  rgba.x = uint8_t(((hiprand_normal(&rngState) + 1.0) * 0.5) * 255);
  rgba.y = uint8_t(((hiprand_normal(&rngState) + 1.0) * 0.5) * 255);
  rgba.z = uint8_t(((hiprand_normal(&rngState) + 1.0) * 0.5) * 255);
  rgba.w = uint8_t(((hiprand_normal(&rngState) + 1.0) * 0.5) * 255);

  surf2Dwrite(rgba, image, x * sizeof(uchar4), y);
}

#endif